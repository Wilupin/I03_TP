#include "hip/hip_runtime.h"
/**
 * \file heat2d_solver_gpu_naive.cu
 * \brief Solve 2D heat equation (finite difference method). GPU version (naive).
 *
 * We solve the 2D Heat equation \f$\partial_t \phi = \alpha \left[
 * \partial^2_x \phi + \partial^2_y \phi \right] \f$, \f$ 0 \leq x
 * \leq L_x \f$, \f$ 0 \leq y \leq L_y \f$, \f$ 0 \leq t\f$.\\
 *
 * Method : Finite Difference, FTCS scheme
 *
 * GPU Features: use only global memory 
 *
 * boundary condition : Dirichlet
 *
 * GPU version : naive
 *
 * \date 17-dec-2009.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sys/time.h> // gettimeofday

// includes, project
#include <helper_functions.h>
#include "CudaTimer.h"
#include "Timer.h"

// parameters + real_t typedef
#include "param.h"

// for output results
#include "output.h"

// GPU solver
#include "heat2d_kernel_gpu_naive.cu"

// CPU solver
#include "heat_kernel_cpu.h"

// initial conditions
#include "misc.h"

// cuda helper
#include "cuda_helper.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv) 
{
  runTest(argc, argv);

  exit(0);
}

////////////////////////////////////////////////////////////////////////////////
//! Run solver on GPU
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv) 
{
  int devID;
  hipDeviceProp_t deviceProps;
  
  devID = findCudaDevice(argc, (const char **)argv);
  
  // get number of SMs on this GPU
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);


  /*
   * read and print parameters
   */
  // default parameter file
  std::string paramFile("heatEqSolver.par");

  // if argv[1] exists use it as a parameter file
  if (argc>1) {
    printf("trying to read parameters from file %s ...\n",argv[1]);
    paramFile = std::string(argv[1]);
  }

  // read parameter file
  readParamFile(paramFile);

  // print parameters on screen
  printParameters("HEAT 2D - GPU (NAIVE)");


  CudaTimer gpuTimer;

  unsigned int mem_size = sizeof(real_t)*NX*NY;

  // allocate host memory
  real_t* data1 = (real_t*) malloc( mem_size);
  real_t* data2 = (real_t*) malloc( mem_size);
  
  ///////////////////////////////////////////////////
  // compute GPU solution to 2D heat equation
  ///////////////////////////////////////////////////
  
  // inital condition
  initCondition2D (data1);
  
  // allocate device memory
  real_t* d_data1;
  real_t* d_data2;
  
  // naive kernel memory allocation (using hipMalloc)
  /* 
   * TODO
   */

  // copy host memory to device
  /*
   * TODO
   */
    
   
  // setup execution parameters for cuda kernel
  // grid dimension for naive kernel
  unsigned int threadsPerBlockX=16;
  unsigned int threadsPerBlockY=16;
  dim3  threads(/* TODO */);
  dim3  grid(/* TODO */);
    
  printf("grid  size : %u %u\n",grid.x,grid.y);
  printf("block size : %u %u\n",threads.x,threads.y);

  // start timer
  gpuTimer.start();

  // time loop executing naive kernel
  int iTime=0;
  int iOutput=-1;
  for (real_t t=0.0f; t<TMAX; t+=(2*DT), iTime+=2) {
    
    if (useOrder2) { // use the 2nd order accurate scheme
      
      heat2d_ftcs_naive_order2_kernel<<< grid, threads >>>( d_data1, d_data2, 
							    NX, NY,
							    o2.R, o2.R2);
      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");
      
      heat2d_ftcs_naive_order2_kernel<<< grid, threads >>>( d_data2, d_data1, 
							    NX, NY,
							    o2.R, o2.R2);   
      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");
      
    } else if (useOrder2b) { // use the 2nd order accurate scheme
      
      heat2d_ftcs_naive_order2b_kernel<<< grid, threads >>>( d_data1, d_data2, 
							     NX, NY,
							     o2.R, o2.R2b);
      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");
      
      heat2d_ftcs_naive_order2b_kernel<<< grid, threads >>>( d_data2, d_data1, 
							     NX, NY,
							     o2.R, o2.R2b);   
      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");

    } else { // use the 4th order accurate scheme
      
      heat2d_ftcs_naive_order4_kernel<<< grid, threads >>>( d_data1, d_data2, 
							    NX, NY,
							    o4.S, o4.S2);
      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");
      
      heat2d_ftcs_naive_order4_kernel<<< grid, threads >>>( d_data2, d_data1, 
							     NX, NY,
							     o4.S, o4.S2);   
      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");
    }

    /* save output (just for cross-checking, do not save when
       measuring computing time */
    if (ENABLE_GPU_SAVE) {

      if (iTime%T_OUTPUT == 0) {
	iOutput++;
	checkCudaErrors( hipMemcpy( /* TODO */ ) );      
      }
      // PGM output
      if (SAVE_PGM and iTime%T_OUTPUT == 0)
	save_pgm(data1, "heat2d_gpu_naive_",iOutput,NX,NY);
      
      // MathGL save (3D view)
      if (SAVE_MGL and iTime%T_OUTPUT == 0)
	save_mgl(data1, "heat2d_gpu_naive_",iOutput,NX,NY);

      // VTK output
      if (SAVE_VTK and iTime%T_OUTPUT == 0)
	save_vtk(data1, "heat2d_gpu_naive_",iOutput);

      // HDF5 output
      if (SAVE_HDF5 and iTime%T_OUTPUT == 0)
	save_hdf5(data1, "heat2d_gpu_naive_",iOutput);

    }

  } // end for loop 
  
  // stop timer
  gpuTimer.stop();

  real_t gpu_time = gpuTimer.elapsed();
  printf( "GPU Processing time: %f (s)\n", gpu_time);
  
  // copy result from device to host
  real_t *resGPU = (real_t*) malloc( mem_size);
  checkCudaErrors( hipMemcpy( /* TODO */ ) );
    
  if (SAVE_HDF5)
      write_xdmf_wrapper("heat2d_gpu_naive",N_ITER,T_OUTPUT);
 
  ////////////////////////////////////////////////////////
  // compute reference (CPU) solution to 2D heat equation
  // for performance comparison
  ////////////////////////////////////////////////////////
  initCondition2D (data1);
  initCondition2D (data2);
  
  Timer cpuTimer;
  cpuTimer.start();
  
  // time loop
  iTime=0;

  for (real_t t=0.0f; t<TMAX; t+=(2*DT), iTime+=2) {
    
    // compute next 2 time steps
    if (useOrder2) {
      heat2d_ftcs_cpu_order2( data1, data2);
      heat2d_ftcs_cpu_order2( data2, data1);
    } else if (useOrder2b) {
      heat2d_ftcs_cpu_order2b( data1, data2);
      heat2d_ftcs_cpu_order2b( data2, data1);
    } else {
      heat2d_ftcs_cpu_order4( data1, data2);
      heat2d_ftcs_cpu_order4( data2, data1);
    }
  }

  // stop timer
  cpuTimer.stop();
  real_t cpu_time = cpuTimer.elapsed();
  
  printf( "CPU Processing time: %g (s)\n", cpu_time);
  printf( "Speedup GPU/CPU : %f\n",cpu_time/gpu_time);

  printf("...comparing the results\n");
  double sum = 0, delta = 0;
  for(unsigned i = 0; i < NX*NY; i++){
    delta += (resGPU[i] - data1[i]) * (resGPU[i] - data1[i]);
    sum   += data1[i] * data1[i];
  }
  double L2norm = sqrt(delta / sum);
  printf("iteration %d relative L2 norm: %E\n", iTime, L2norm);
 
  // cleanup memory
  free(data1);
  free(data2);
  free(resGPU);
  
  checkCudaErrors(hipFree(d_data1));
  checkCudaErrors(hipFree(d_data2));
  
  hipDeviceSynchronize();
  hipDeviceReset();

  exit(0);
}
